#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"


namespace caffe {

template <typename Dtype>
__global__ void AdamWRUpdate(int N, Dtype* theta, Dtype* g, Dtype* m, Dtype* v,
    Dtype beta1, Dtype beta2, Dtype eps_hat, Dtype corrected_local_rate, Dtype local_decay, Dtype yita) {
  CUDA_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float mi = m[i] = m[i]*beta1 + gi*(1-beta1);
    float vi = v[i] = v[i]*beta2 + gi*gi*(1-beta2);
    g[i] = yita * ((corrected_local_rate * mi / (sqrt(vi) + eps_hat)) + local_decay * theta[i]);
    // g[i] = corrected_local_rate * mi / (sqrt(vi) + eps_hat);
  }
}
template <typename Dtype>
void adamwr_update_gpu(int N, Dtype *theta, Dtype* g, Dtype* m, Dtype* v, Dtype beta1,
    Dtype beta2, Dtype eps_hat, Dtype corrected_local_rate, Dtype local_decay, Dtype yita) {

  AdamWRUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, theta, g, m, v, beta1, beta2, eps_hat, corrected_local_rate, local_decay, yita);
  CUDA_POST_KERNEL_CHECK;
}

template void adamwr_update_gpu<float>(int, float*, float*, float*, float*,
    float, float, float, float, float, float);
template void adamwr_update_gpu<double>(int, double*, double*, double*, double*,
    double, double, double, double, double, double);

}  // namespace caffe